
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){
	hipDeviceProp_t devProps;
	hipGetDeviceProperties(&devProps,0);
	printf("Device 0 name: %s\n",devProps.name);
	printf("Compute capability %d.%d\n",devProps.major,devProps.minor);
}
